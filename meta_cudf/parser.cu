//#include "opt1/meta_def.cuh"
#include <fstream>
#include <memory>

#include <boost/mp11.hpp>
#include <cudf/io/types.hpp>
#include <rmm/device_buffer.hpp>
#include <rmm/device_scalar.hpp>
#include <thrust/logical.h>
#include <iomanip>
#include <meta_json_parser/parser_output_device.cuh>
#include <meta_json_parser/parser_kernel.cuh>
#include <meta_json_parser/action/jstring.cuh>

#include <meta_def.cuh>

using namespace std;
using namespace boost::mp11;

enum class end_of_line {
    unknown,
    uniks, //< LF, or "\n": end-of-line convention used by Unix
    win   //< CRLF, or "\r\n": end-of-line convention used by MS Windows
};

namespace EndOfLine
{
    struct Unix {};
    struct Win {};
}

struct NoError
{
    __device__ __host__ bool operator()(ParsingError e)
    {
        return ParsingError::None == e;
    }
};

template<class EndOfLineT>
struct LineEndingHelper
{
private:
    __device__ __forceinline__ static void error() { assert("Unknown end of line."); }
public:
    __device__ __forceinline__ static uint32_t get_mask(const uint32_t& val) { error(); return 0; }
    __device__ __forceinline__ static bool is_newline(const uint32_t& val) { error(); return false; }
    __device__ __forceinline__ static uint32_t eol_length() { error(); return 0; }
};

template<>
struct LineEndingHelper<EndOfLine::Unix>
{
    __device__ __forceinline__ static uint32_t get_mask(const uint32_t& val)
    {
        return __vcmpeq4(val, '\n\n\n\n');
    }
    __device__ __forceinline__ static bool is_newline(const uint32_t& val)
    {
        return get_mask(val);
    }
    __device__ __forceinline__ static constexpr uint32_t eol_length()
    {
        return 1;
    }
};

/// <summary>
/// Implemented with assumption that \r can only be found right before \n
/// </summary>
template<>
struct LineEndingHelper<EndOfLine::Win>
{
    __device__ __forceinline__ static uint32_t get_mask(const uint32_t& val)
    {
        return __vcmpeq4(val, '\r\r\r\r');
    }
    __device__ __forceinline__ static bool is_newline(const uint32_t& val)
    {
        return get_mask(val);
    }
    __device__ __forceinline__ static constexpr uint32_t eol_length()
    {
        return 2;
    }
};

template<class EndOfLineT>
struct IsNewLine
{
    __device__ __forceinline__ bool operator()(const hipcub::KeyValuePair<ptrdiff_t, uint32_t> c) const {
        return LineEndingHelper<EndOfLineT>::is_newline(c.value);
    }
};

template<class EndOfLineT>
class OutputIndicesIterator
{
public:

    // Required iterator traits
    typedef OutputIndicesIterator<EndOfLineT>            self_type;              ///< My own type
    typedef ptrdiff_t                                    difference_type;        ///< Type to express the result of subtracting one iterator from another
    typedef hipcub::KeyValuePair<difference_type, uint32_t> value_type;             ///< The type of the element the iterator can point to
    typedef value_type*                                  pointer;                ///< The type of a pointer to an element the iterator can point to
    typedef value_type                                   reference;              ///< The type of a reference to an element the iterator can point to

#if (THRUST_VERSION >= 100700)
    // Use Thrust's iterator categories so we can use these iterators in Thrust 1.7 (or newer) methods
    typedef typename thrust::detail::iterator_facade_category<
        thrust::any_system_tag,
        thrust::random_access_traversal_tag,
        value_type,
        reference
    >::type iterator_category;                                        ///< The iterator category
#else
    typedef std::random_access_iterator_tag     iterator_category;      ///< The iterator category
#endif  // THRUST_VERSION

private:

    InputIndex*  itr;

public:

    /// Constructor
    __host__ __device__ __forceinline__ OutputIndicesIterator(InputIndex* itr) : itr(itr) {}

    /// Assignment operator
    __device__ __forceinline__ self_type& operator=(const value_type &val)
    {
        int inner_offset = LineEndingHelper<EndOfLineT>::eol_length();
        //undefined behavior for 2 byte jsons. e.g. \n[]\n or \n{}\n
        uint32_t mask = LineEndingHelper<EndOfLineT>::get_mask(val.value);
        switch (mask)
        {
            case 0xFF'00'00'00u:
                inner_offset += 3;
                break;
            case 0x00'FF'00'00u:
                inner_offset += 2;
                break;
            case 0x00'00'FF'00u:
                inner_offset += 1;
                break;
            case 0x00'00'00'FFu:
                //inner_offset += 0;
                break;
            default:
                break;
        }
        *itr = static_cast<InputIndex>(val.key * 4) + inner_offset;
        return *this;
    }

    /// Array subscript
    template <typename Distance>
    __host__ __device__ __forceinline__ self_type operator[](Distance n)
    {
        self_type offset = OutputIndicesIterator(itr + n);
        return offset;
    }
};

struct benchmark_input
{
    vector<char> data;
    int count;
    end_of_line eol;
    int bytes_per_string;
};

struct benchmark_device_buffers
{
    ParserOutputDevice<BaseAction> parser_output_buffers;
    rmm::device_uvector<char> readonly_buffers;
    rmm::device_uvector<char> input_buffer;
    rmm::device_uvector<InputIndex> indices_buffer;
    rmm::device_uvector<ParsingError> err_buffer;
    rmm::device_uvector<void*> output_buffers;
    int count;

    vector<void*> host_output_buffers;

    benchmark_device_buffers(ParserOutputDevice<BaseAction>&& parser_output_buffers, rmm::device_uvector<char>&& readonly_buffers,
                             rmm::device_uvector<char>&& input_buffer, rmm::device_uvector<InputIndex> indices_buffer,
                             rmm::device_uvector<ParsingError>&& err_buffer, rmm::device_uvector<void*> output_buffers,
                             vector<void*> host_output_buffers, int count):
                             parser_output_buffers(std::move(parser_output_buffers)), readonly_buffers(std::move(readonly_buffers)),
                             input_buffer(std::move(input_buffer)), indices_buffer(std::move(indices_buffer)), err_buffer(std::move(err_buffer)),
                             output_buffers(std::move(output_buffers)), host_output_buffers(std::move(host_output_buffers)), count(count)
    {}
};

benchmark_input get_input(const char* filename, int input_count);
KernelLaunchConfiguration prepare_dynamic_config(benchmark_input& input);
shared_ptr<benchmark_device_buffers> initialize_buffers(benchmark_input& input, KernelLaunchConfiguration* conf, rmm::cuda_stream_view stream, rmm::mr::device_memory_resource* mr);
end_of_line detect_eol(benchmark_input& input);
void launch_kernel(shared_ptr<benchmark_device_buffers> device_buffers, rmm::cuda_stream_view stream, rmm::mr::device_memory_resource* mr);

template<class EndOfLineT>
void find_newlines(rmm::device_uvector<char>& d_input, size_t input_size, rmm::device_uvector<InputIndex>& d_indices, int count, rmm::cuda_stream_view stream, rmm::mr::device_memory_resource* mr)
{
    d_indices.set_element_to_zero_async(0, stream); //Skopiowanie pierwszego indeksu ograniczającego linie, czyli 0

    hipcub::ArgIndexInputIterator<uint32_t*> arg_iter(reinterpret_cast<uint32_t*>(d_input.data()));
    OutputIndicesIterator<EndOfLineT> out_iter(d_indices.data() + 1); // +1, we need to add 0 at index 0

    size_t temp_storage_bytes = 0;
    auto d_num_selected = rmm::device_scalar<int>(stream, mr);

    hipcub::DeviceSelect::If(
        nullptr,
        temp_storage_bytes,
        arg_iter,
        out_iter,
        d_num_selected.data(),
        (input_size + 3) / 4,
        IsNewLine<EndOfLineT>(),
        stream
    );

    auto d_temp_storage = rmm::device_buffer(temp_storage_bytes, stream, mr);

    hipcub::DeviceSelect::If(
        d_temp_storage.data(),
        temp_storage_bytes,
        arg_iter,
        out_iter,
        d_num_selected.data(),
        (input_size + 3) / 4,
        IsNewLine<EndOfLineT>(),
        stream
    );

//    // Following lines could be commented out as it is only validation step
//    hipStreamSynchronize(stream);
//    int h_num_selected = -1;
//    hipMemcpy(&h_num_selected, d_num_selected, sizeof(int), hipMemcpyDeviceToHost);
//    if (h_num_selected != count)
//    {
//        cout << "Found " << h_num_selected << " new lines instead of declared " << count << ".\n";
//        throw runtime_error("Invalid number of new lines.");
//    }

}

cudf::io::table_with_metadata generate_example_metadata(const char* filename, int count) {
//	hipStreamCreate(&stream);
    rmm::hip_stream stream;
    rmm::mr::device_memory_resource* mr = rmm::mr::get_current_device_resource();

    auto input = get_input(filename, count);

    KernelLaunchConfiguration conf = prepare_dynamic_config(input);
    shared_ptr<benchmark_device_buffers> device_buffers = initialize_buffers(input, &conf, stream, mr);
    launch_kernel(device_buffers, stream, mr);
    auto cudf_table  = device_buffers->parser_output_buffers.ToCudf(stream, mr);

    vector<cudf::io::column_name_info> column_names(cudf_table.num_columns());

    generate(column_names.begin(), column_names.end(), [i = 1]() mutable {
        return cudf::io::column_name_info("Column " + to_string(i++));
    });

    cudf::io::table_metadata metadata{column_names};

    return cudf::io::table_with_metadata{
        make_unique<cudf::table>(cudf_table),
        metadata
    };
}

void launch_kernel(shared_ptr<benchmark_device_buffers> device_buffers, rmm::cuda_stream_view stream, rmm::mr::device_memory_resource* mr)
{
    using GroupSize = WorkGroupSize;
    constexpr int GROUP_SIZE = WorkGroupSize::value;
    constexpr int GROUP_COUNT = 1024 / GROUP_SIZE;
    using GroupCount = mp_int<GROUP_COUNT>;

    using RT = RuntimeConfiguration<GroupSize, GroupCount>;
    using PC = ParserConfiguration<RT, BaseAction>;
    using PK = ParserKernel<PC>;

    PK pk(device_buffers->parser_output_buffers.m_launch_config, stream, mr);

    pk.Run(
        device_buffers->input_buffer.data(),
        device_buffers->indices_buffer.data(),
        device_buffers->err_buffer.data(),
        device_buffers->output_buffers.data(),
        device_buffers->count,
        device_buffers->host_output_buffers.data()
    );
}

end_of_line detect_eol(benchmark_input& input)
{
    auto found = std::find_if(input.data.begin(), input.data.end(), [](char& c) {
        return c == '\r' || c == '\n';
    });
    if (found == input.data.end())
        return end_of_line::unknown;
    if (*found == '\n')
        return end_of_line::uniks;
    // *found == '\r'
    if ((found + 1) == input.data.end() || *(found + 1) != '\n')
        return end_of_line::unknown;
    return end_of_line::win;
}

KernelLaunchConfiguration prepare_dynamic_config(benchmark_input& input)
{
    KernelLaunchConfiguration conf;

    using DynamicStringActions = mp_copy_if_q<
        ActionIterator<BaseAction>,
        mp_bind<
            mp_similar,
            JStringDynamicCopy<void>,
            _1
        >
    >;

    using DynamicStringActionsV2 = mp_copy_if_q<
        ActionIterator<BaseAction>,
        mp_bind<
            mp_similar,
            JStringDynamicCopyV2<void>,
            _1
        >
    >;

    using DynamicStringActionsV3 = mp_copy_if_q<
        ActionIterator<BaseAction>,
        mp_bind<
            mp_similar,
            JStringDynamicCopyV3<void>,
            _1
        >
    >;

    mp_for_each<
        mp_append<
            DynamicStringActions,
            DynamicStringActionsV2
        >
    >([&conf, &input](auto a) {
        using Action = decltype(a);
        using Tag = typename Action::DynamicStringRequestTag;
        conf.SetDynamicSize<BaseAction, Tag>(input.bytes_per_string);
    });

    mp_for_each<DynamicStringActionsV3>([&conf, &input](auto a) {
        using Action = decltype(a);
        using TagInternal = typename Action::DynamicStringInternalRequestTag;
        conf.SetDynamicSize<BaseAction, TagInternal>(input.bytes_per_string);
        using Tag = typename Action::DynamicStringRequestTag;
        conf.SetDynamicSize<BaseAction, Tag>(input.bytes_per_string);
    });

    return std::move(conf);
}

shared_ptr<benchmark_device_buffers> initialize_buffers(benchmark_input& input, KernelLaunchConfiguration* conf, rmm::cuda_stream_view stream, rmm::mr::device_memory_resource* mr)
{
    using GroupSize = WorkGroupSize;
    constexpr int GROUP_SIZE = WorkGroupSize::value;
    constexpr int GROUP_COUNT = 1024 / GROUP_SIZE;
    using GroupCount = mp_int<GROUP_COUNT>;
    using RT = RuntimeConfiguration<GroupSize, GroupCount>;
    using PC = ParserConfiguration<RT, BaseAction>;
    using PK = ParserKernel<PC>;
    using M3 = typename PK::M3;
    using BUF = typename M3::ReadOnlyBuffer;
    using KC = typename PK::KC;
    using OM = typename KC::OM;
    constexpr size_t REQUEST_COUNT = boost::mp11::mp_size<typename OutputConfiguration<BaseAction>::RequestList>::value;


    auto result = make_shared<benchmark_device_buffers>(
            ParserOutputDevice<BaseAction>(conf, input.count, stream, mr),
            rmm::device_uvector<char>(sizeof(BUF), stream, mr),
            rmm::device_uvector<char>(input.data.size(), stream, mr),
            rmm::device_uvector<InputIndex>(input.count + 1, stream, mr),
            rmm::device_uvector<ParsingError>(input.count, stream, mr),
            rmm::device_uvector<void*>(REQUEST_COUNT, stream, mr),
            vector<void*>(REQUEST_COUNT),
            input.count
    );

    for (int i = 0; i < REQUEST_COUNT; ++i)
    {
        result->host_output_buffers[i] = result->parser_output_buffers.m_d_outputs[i]->data();
    }

    hipMemcpyAsync(result->input_buffer.data(), input.data.data(), input.data.size(), hipMemcpyHostToDevice, stream);
    hipMemcpyAsync(result->output_buffers.data(), result->host_output_buffers.data(), sizeof(void*) * REQUEST_COUNT, hipMemcpyHostToDevice, stream);

    //End of line might be passed as an option to the program
    if (input.eol == end_of_line::unknown)
        input.eol = detect_eol(input);

    switch (input.eol)
    {
        case end_of_line::uniks:
            find_newlines<EndOfLine::Unix>
                (result->input_buffer, input.data.size(), result->indices_buffer, input.count, stream, mr);
            break;
        case end_of_line::win:
            find_newlines<EndOfLine::Win>
                (result->input_buffer, input.data.size(), result->indices_buffer, input.count, stream, mr);
            break;
        case end_of_line::unknown:
        default:
            std::cerr << "Unknown end of line character!";
            throw std::runtime_error("Unknown end of line character");
    }

//    auto test = vector<uint32_t>(2001);
//    hipMemcpy(test.data(), result.indices_buffer, 2001, hipMemcpyDeviceToHost);

    //Dzieki sprytnej implementacji iteratora, result.indices_buffer zawiera indeksy charakterow a nie uint32_t,
    //dokladnie sa to indeksy nowych linii

    return result;
}

benchmark_input get_input(const char* filename, int input_count)
{
    ifstream file(filename, ifstream::ate | ifstream::binary);
    if (!file.good())
    {
        cout << "Error reading file \"" << filename << "\".\n";
        throw std::runtime_error("Error reading file.");
    }
    vector<char> data(file.tellg());
    file.seekg(0);
    file.read(data.data(), static_cast<streamsize>(data.size()));

    return benchmark_input
        {
            std::move(data),
            input_count,
            end_of_line::unknown,
            32
        };
}


